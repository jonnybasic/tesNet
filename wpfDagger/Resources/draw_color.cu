#include "hip/hip_runtime.h"

#include "common.h"

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtBuffer<uchar4, 2>   output_buffer;

rtDeclareVariable(float3, draw_color, , );

RT_PROGRAM void draw_solid_color()
{
	//result_buffer[launch_index] = make_float4(draw_color, 0.f);
	output_buffer[launch_index] = make_color(draw_color);
}
