#include "hip/hip_runtime.h"
#include "common_material.h"
#include "random.h"

#if 0

//-----------------------------------------------------------------------------
//
//  Lambertian surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, diffuse_color, , "Diffuse color assigned to the material");

RT_PROGRAM void diffuse()
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -current_ray.direction, world_geometric_normal);

	//float3 hitpoint = current_ray.origin + hit_distance * current_ray.direction;

	//
	// Generate a reflection current_ray.  This will be traced back in current_ray-gen.
	//
	prd_radiance.origin = hit_position;

	float z1 = rnd(prd_radiance.seed);
	float z2 = rnd(prd_radiance.seed);
	float3 p;
	cosine_sample_hemisphere(z1, z2, p);
	optix::Onb onb(ffnormal);
	onb.inverse_transform(p);
	prd_radiance.direction = p;

	// NOTE: f/pdf = 1 since we are perfectly importance sampling lambertian
	// with cosine density.
	prd_radiance.attenuation = prd_radiance.attenuation * diffuse_color;
	prd_radiance.countEmitted = false;

	//
	// Next event estimation (compute direct lighting).
	//
	unsigned int num_lights = lights.size();
	float3 result = make_float3(0.0f);

	for (int i = 0; i < num_lights; ++i)
	{
		// Choose random point on light
		ParallelogramLight light = lights[i];
		const float z1 = rnd(current_prd.seed);
		const float z2 = rnd(current_prd.seed);
		const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

		// Calculate properties of light sample (for area based pdf)
		const float  Ldist = length(light_pos - hitpoint);
		const float3 L = normalize(light_pos - hitpoint);
		const float  nDl = dot(ffnormal, L);
		const float  LnDl = dot(light.normal, L);

		// cast shadow ray
		if (nDl > 0.0f && LnDl > 0.0f)
		{
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.inShadow = false;
			// Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
			Ray shadow_ray = make_Ray(hitpoint, L, SHADOW_RAY_TYPE, scene_epsilon, Ldist - scene_epsilon);
			rtTrace(top_object, shadow_ray, shadow_prd);

			if (!shadow_prd.inShadow)
			{
				const float A = length(cross(light.v1, light.v2));
				// convert area based pdf to solid angle
				const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
				result += light.emission * weight;
			}
		}
	}

	prd_radiance.radiance = result;
}

#endif

rtDeclareVariable(float3, Kd, , ) = { 0.9f, 0.9f, 0.9f };
rtDeclareVariable(float3, geometry_color, , ) = { 1.0f, 1.0f, 1.0f };

rtDeclareVariable(float4, light_direction, , ) = { -1.0f, -0.75f, 0.5f };
rtDeclareVariable(float3, light_color, , ) = { 1.0f, 1.0f, 1.0f };
rtDeclareVariable(float3, light_v0, , ) = { 1.0f, 0.0f, 0.0f };
rtDeclareVariable(float3, light_v1, , ) = { 0.0f, 1.0f, 0.0f };
//rtBuffer<DirectionalLight> light_buffer;

RT_PROGRAM void any_hit_shadow()
{
	prd_shadow.attenuation = make_float3(0.0f);
	rtTerminateRay();
}

// Note: both the hemisphere and direct light sampling below use pure random numbers to avoid any patent issues.
// Stratified sampling or QMC would improve convergence.  Please keep this in mind when judging noise levels.

RT_PROGRAM void closest_hit_white()
{
    prd_radiance.result = make_float3(1.0f);
}

RT_PROGRAM void closest_hit_radiance()
{
	const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	const float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	const float3 ffnormal = faceforward(world_shading_normal, -current_ray.direction, world_geometric_normal);

	const float z1 = rnd(prd_radiance.seed);
	const float z2 = rnd(prd_radiance.seed);

	float3 w_in;
	cosine_sample_hemisphere(z1, z2, w_in);
	const Onb onb(ffnormal);
	onb.inverse_transform(w_in);
	const float3 fhp = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_position);

	prd_radiance.origin = front_hit_position;
	prd_radiance.direction = w_in;
	prd_radiance.attenuation *= Kd * geometry_color;

	// Add direct light sample weighted by shadow term and 1/probability.
	// The pdf for a directional area light is 1/solid_angle.

	const float3 light_center = fhp + make_float3(light_direction.x);
	const float light_radius = light_direction.w;
	const float r1 = rnd(prd_radiance.seed);
	const float r2 = rnd(prd_radiance.seed);
	const float2 disk_sample = square_to_disk(make_float2(r1, r2));
	const float3 jittered_pos = light_center
		+ light_radius * disk_sample.x * light_v0
		+ light_radius * disk_sample.y * light_v1;
	const float3 L = normalize(jittered_pos - fhp);

	const float NdotL = dot(ffnormal, L);
	if (NdotL > 0.0f)
	{
		PerRayData_shadow shadow_prd;
		shadow_prd.attenuation = make_float3(1.0f);
		Ray shadow_ray(fhp, L, SHADOW_RAY_TYPE, 0.0f);
		rtTrace(top_object, shadow_ray, shadow_prd);

		const float solid_angle = light_radius * light_radius * M_PIf;
		prd_radiance.radiance += NdotL * light_color * solid_angle * shadow_prd.attenuation;
	}
}
