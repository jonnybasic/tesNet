#include "hip/hip_runtime.h"
//#include "common.h"
//#include <optixu/optixu_aabb_namespace.h>
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float3, geometric_normal,   attribute geometric_normal,   "normal to the geometry");
rtDeclareVariable(float3, shading_normal,     attribute shading_normal,     "normal used for shading");
rtDeclareVariable(float3, texcoord,           attribute texcoord,           "polar coordinates");
//rtDeclareVariable(float3, front_hit_position, attribute front_hit_position, "intersection front position");
//rtDeclareVariable(float3, back_hit_position,  attribute back_hit_position,  "intersection back position");

rtDeclareVariable(Ray, current_ray, rtCurrentRay, "built-in access to ray");
rtDeclareVariable(float, scene_epsilon, , );

// Adapted from NVIDIA to be a unit sphere at the origin
template<bool use_robust_method>
static __device__ void intersect_single(void)
{
	static const float radius = 1.0f;

	float3 O = current_ray.origin;
	float  l = 1 / length(current_ray.direction);
	float3 D = current_ray.direction * l;
	float b = dot(O, D);
    // radius*radius = radius
	float c = dot(O, O) - radius;
	float disc = b * b - c;
	if (disc > 0.0f)
	{
		float sdisc = sqrtf(disc);
		float root1 = (-b - sdisc);
		bool do_refine = false;
		float root11 = 0.0f;
        // 10.0f*radius = 10.0f
		if (use_robust_method && fabsf(root1) > 10.0f)
		{
			do_refine = true;
		}
		if (do_refine)
		{
			float3 O1 = O + root1 * D;
			b = dot(O1, D);
            // radius*radius = radius
			c = dot(O1, O1) - radius;
			disc = b * b - c;
			if (disc > 0.0f)
			{
				rtPrintf("us: refined\n");
				sdisc = sqrtf(disc);
				root11 = (-b - sdisc);
			}
		}
		bool second_check = true;
		if (rtPotentialIntersection((root1 + root11) * l))
		{
			const float3 hit_position = (O + (root1 + root11) * D);
			shading_normal = geometric_normal = hit_position;
			//const float3 offset = shading_normal * scene_epsilon;
			//front_hit_position = hit_position + offset;
			//back_hit_position = hit_position - offset;
			float3 polar = cart_to_pol(geometric_normal);
			texcoord = make_float3(polar.x * 0.5f * M_1_PIf,
				                  (polar.y + M_PI_2f) * M_1_PIf,
                                   polar.z);
			rtPrintf("us: outside -> hit inside: d=%f, s=%f, t=%f\n",
					 ((root1 + root11) * l), texcoord.x, texcoord.y);
			if (rtReportIntersection(0))
			{
				second_check = false;
			}
		}
		if (second_check)
		{
			//float root2 = (-b + sdisc);
            float root2 = (-b + sdisc) + (do_refine ? root1 : 0);
			if (rtPotentialIntersection(root2 * l))
			{
				//const float3 hit_position = (O + (root1 + root11) * D);
                const float3 hit_position = (O + root2 * D);
				shading_normal = geometric_normal = hit_position;
				//const float3 offset = shading_normal * scene_epsilon;
				//front_hit_position = hit_position - offset;
				//back_hit_position = hit_position + offset;
				float3 polar = cart_to_pol(geometric_normal);
				texcoord = make_float3(polar.x * 0.5f * M_1_PIf,
					                  (polar.y + M_PI_2f) * M_1_PIf,
                                       polar.z);
				rtPrintf("us: inside -> hit outside: d=%f, s=%f, t=%f\n",
						 (root2 * l), texcoord.x, texcoord.y);
				rtReportIntersection(0);
			}
		}
	}
}

RT_PROGRAM void intersect(int primIdx)
{
	intersect_single<true>();
}

RT_PROGRAM void intersect_fast(int primInx)
{
	intersect_single<false>();
}

RT_PROGRAM void bounds(int primIdx, float results[6])
{
	static const float radius = 1.0f;
	// A unit sphere with a radius of 1.0
	Aabb* aabb = (Aabb*)results;
	aabb->m_min = make_float3(-radius);
	aabb->m_max = make_float3(radius);
}
