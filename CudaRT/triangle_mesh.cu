#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "common.h"
#include <optixu/optixu_aabb_namespace.h>

rtDeclareVariable(float2, texcoord,           attribute texcoord,           "texture coordinates");
rtDeclareVariable(float3, geometric_normal,   attribute geometric_normal,   "normal to the geometry");
rtDeclareVariable(float3, shading_normal,     attribute shading_normal,     "normal used for shading");
rtDeclareVariable(float3, front_hit_position, attribute front_hit_position, "intersection front position");
rtDeclareVariable(float3, back_hit_position,  attribute back_hit_position,  "intersection back position");

rtDeclareVariable(Ray, current_ray,    rtCurrentRay,            "built-in access to ray");

// This is to be plugged into an RTgeometry object to represent
// a triangle mesh with a vertex buffer of triangle soup (triangle list)
// with an interleaved position, normal, texturecoordinate layout.

rtBuffer<float3> vertex_buffer;
rtBuffer<float3> normal_buffer;
rtBuffer<float2> texcoord_buffer;
rtBuffer<int3>   index_buffer;
//rtBuffer<int>    material_buffer;

static __device__ void intersect_primitive(int primIdx)
{
	const int3 v_idx = index_buffer[primIdx];

	const float3 p0 = vertex_buffer[v_idx.x];
	const float3 p1 = vertex_buffer[v_idx.y];
	const float3 p2 = vertex_buffer[v_idx.z];

	// Intersect ray with triangle
	float3 n;
	float  t, beta, gamma;
	if (intersect_triangle(current_ray, p0, p1, p2, n, t, beta, gamma))
	{
		if (rtPotentialIntersection(t))
		{
			geometric_normal = normalize(n);
			if (normal_buffer.size() == 0)
			{
				shading_normal = geometric_normal;
			}
			else
			{
				float3 n0 = normal_buffer[v_idx.x];
				float3 n1 = normal_buffer[v_idx.y];
				float3 n2 = normal_buffer[v_idx.z];
				shading_normal = normalize(n1 * beta + n2 * gamma + n0 * (1.0f - beta - gamma));
			}
			if (texcoord_buffer.size() == 0)
			{
				texcoord = make_float2(0.0f, 0.0f);
			}
			else
			{
				float2 t0 = texcoord_buffer[v_idx.x];
				float2 t1 = texcoord_buffer[v_idx.y];
				float2 t2 = texcoord_buffer[v_idx.z];
				texcoord = (t1 * beta + t2 * gamma + t0 * (1.0f - beta - gamma));
			}
			const float3 hit_position = current_ray.origin + t * current_ray.direction;
			const float3 offset = geometric_normal * scene_epsilon;
			front_hit_position = hit_position + offset;
			back_hit_position = hit_position - offset;

			//rtReportIntersection(material_buffer[primIdx]);
			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void intersect(int primIdx)
{
	intersect_primitive(primIdx);
}

RT_PROGRAM void bounds(int primIdx, float result[6])
{
	const int3 v_idx = index_buffer[primIdx];

	const float3 v0 = vertex_buffer[v_idx.x];
	const float3 v1 = vertex_buffer[v_idx.y];
	const float3 v2 = vertex_buffer[v_idx.z];
	const float  area = length(cross(v1 - v0, v2 - v0));

	Aabb* aabb = (Aabb*)result;

	if (area > 0.0f && !isinf(area))
	{
		aabb->m_min = fminf(fminf(v0, v1), v2);
		aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
	}
	else
	{
		aabb->invalidate();
	}
}
