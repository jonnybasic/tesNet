#include "hip/hip_runtime.h"
#include "common.h"

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , ) = { 1.0f, 0.0f, 1.0f };

rtBuffer<uchar4, 2> output_buffer;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );

RT_PROGRAM void pinhole_camera()
{
    float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
    float3 ray_origin = eye;
    float3 ray_direction = normalize(d.x*U + d.y*V + W);

    Ray ray = make_Ray(ray_origin,
                       ray_direction,
                       RADIANCE_RAY_TYPE,
                       scene_epsilon,
                       RT_DEFAULT_MAX);

    PerRayData_radiance prd;
    prd.depth = 0;
    prd.done = false;
    prd.result = make_float3(0.0f);

    // brdf attenuation from surface interaction
    prd.attenuation = make_float3(1.0f);
    // light from a light source or miss program
    prd.radiance = make_float3(0.0f);

    rtTrace(top_object, ray, prd);

    output_buffer[launch_index] = make_color(prd.result);
}

RT_PROGRAM void exception()
{
    rtPrintExceptionDetails();
    output_buffer[launch_index] = make_color(bad_color);
}
